#include "hip/hip_runtime.h"
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
 * Created by: Hang Zhang
 * ECE Department, Rutgers University
 * Email: zhang.hang@rutgers.edu
 * Copyright (c) 2016
 *
 * Feel free to reuse and distribute this software for research or 
 * non-profit purpose, subject to the following conditions:
 *  1. The code must retain the above copyright notice, this list of
 *     conditions.
 *  2. Original authors' names are not deleted.
 *  3. The authors' names are not used to endorse or promote products
 *      derived from this software 
 *+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
 */
#include "HZPROC.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"

#ifdef __cplusplus
extern "C" {
#endif

__global__ void HZMapping_kernel (
	THCDeviceTensor<float, 3> input,
	THCDeviceTensor<float, 3> output,
	THCDeviceTensor<float, 3> table)
{
  /* declarations of the variables */
  int ch, xo, yo, xi, yi, inwidth, inheight;
  /* Get the index and channels */ 
  ch = blockIdx.z;
  xo = blockIdx.x * blockDim.x + threadIdx.x;
  yo = blockIdx.y * blockDim.y + threadIdx.y;
	/* boundary check for output */
	if (xo >= output.getSize(2), yo >= output.getSize(1))	return;
	inwidth  = input.getSize(2);
	inheight = input.getSize(1);
	/* main operation */
	xi = table[0][yo][xo];
	yi = table[1][yo][xo];
	if(xi >= 0 && xi < inwidth && yi >-0 && yi < inheight)
		output[ch][yo][xo] = input[ch][yi][xi].ldg();
	else
		output[ch][yo][xo] = 0;
}

void HZMapping(THCState *state, THCudaTensor *input_, THCudaTensor *output_,
							THCudaTensor *table_)
/*
 * mapping the image pixels based on the lookuptable
 */
{
	/* Check the GPU index */
	HZPROC_assertSameGPU(state, 3, input_, output_, table_);
	/* Device tensors */
	THCDeviceTensor<float, 3> input  = devicetensor<3>(state, input_);
	THCDeviceTensor<float, 3> output = devicetensor<3>(state, output_);
	THCDeviceTensor<float, 3> table  = devicetensor<3>(state, table_);
	/* kernel function */
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 threads(16, 16);
	dim3 blocks(output.getSize(2)/16+1, output.getSize(1)/16+1, 
							output.getSize(0));
	
	HZMapping_kernel<<<blocks, threads, 0, stream>>>(input, output, table);
	THCudaCheck(hipGetLastError());
}

#ifdef __cplusplus
}
#endif
